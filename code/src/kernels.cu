#include "hip/hip_runtime.h"
#include "../include/Point.h"
#include <hip/hip_runtime.h>
#include <math.h>

/**
 * @brief CUDA kernel for bilinear interpolation
 * 
 * @param grid Input elevation grid (flattened 2D array)
 * @param points Input query points
 * @param results Output interpolated values
 * @param num_points Number of points to interpolate
 * @param min_lon Minimum longitude value
 * @param max_lon Maximum longitude value
 * @param min_lat Minimum latitude value
 * @param max_lat Maximum latitude value
 * @param num_lon Number of grid points in longitude direction
 * @param num_lat Number of grid points in latitude direction
 * @param lon_step Grid cell size in longitude direction
 * @param lat_step Grid cell size in latitude direction
 */
__global__ void bilinearInterpolationKernel(
    const double* __restrict__ grid,
    const Point* __restrict__ points,
    double* __restrict__ results,
    int num_points,
    double min_lon, double max_lon,
    double min_lat, double max_lat,
    int num_lon, int num_lat,
    double lon_step, double lat_step
) {
    // Calculate global thread ID
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if this thread should process a point
    if (tid < num_points) {
        // Get the point to interpolate
        double lon = points[tid].lon;
        double lat = points[tid].lat;
        
        // Check if the point is inside the grid bounds
        if (lon < min_lon || lon > max_lon || lat < min_lat || lat > max_lat) {
            results[tid] = NAN;  // Return NaN for out-of-bounds points
            return;
        }
        
        // Find the indices of the grid cell that contains the point
        double x = (lon - min_lon) / lon_step;  // Normalized x-coordinate
        double y = (lat - min_lat) / lat_step;  // Normalized y-coordinate
        
        int x0 = floor(x);              // Lower x index
        int y0 = floor(y);              // Lower y index
        int x1 = min(x0 + 1, num_lon - 1);  // Upper x index (boundary check)
        int y1 = min(y0 + 1, num_lat - 1);  // Upper y index (boundary check)
        
        // Calculate interpolation weights
        double wx = x - x0;  // Weight for x interpolation
        double wy = y - y0;  // Weight for y interpolation
        
        // Get the four corner elevations
        // Grid is stored in row-major order (y is the outer index)
        double z00 = grid[y0 * num_lon + x0];  // Bottom-left elevation
        double z01 = grid[y0 * num_lon + x1];  // Bottom-right elevation
        double z10 = grid[y1 * num_lon + x0];  // Top-left elevation
        double z11 = grid[y1 * num_lon + x1];  // Top-right elevation
        
        // Perform bilinear interpolation
        // First interpolate along x direction for both y values
        double z0 = (1 - wx) * z00 + wx * z01;  // Bottom edge interpolation
        double z1 = (1 - wx) * z10 + wx * z11;  // Top edge interpolation
        
        // Then interpolate along y direction
        results[tid] = (1 - wy) * z0 + wy * z1;  // Final interpolated value
    }
}

/**
 * @brief Device function for cubic interpolation using the Catmull-Rom formula.
 * 
 * @param p0 Value at x = -1
 * @param p1 Value at x =  0
 * @param p2 Value at x = +1
 * @param p3 Value at x = +2
 * @param t  Fractional distance between p1 and p2
 * @return double Interpolated value
 */
__device__ double cubicInterpolate(double p0, double p1, double p2, double p3, double t) {
    return 0.5 * (2.0 * p1 +
                  (-p0 + p2) * t +
                  (2.0 * p0 - 5.0 * p1 + 4.0 * p2 - p3) * t * t +
                  (-p0 + 3.0 * p1 - 3.0 * p2 + p3) * t * t * t);
}

/**
 * @brief CUDA kernel for cubic (bicubic) interpolation.
 * 
 * Computes the interpolated value using a 4x4 neighborhood and 
 * the Catmull-Rom cubic interpolation in both x and y directions.
 * 
 * @param grid Input elevation grid (flattened 2D array)
 * @param points Input query points
 * @param results Output interpolated values
 * @param num_points Number of points to interpolate
 * @param min_lon Minimum longitude value
 * @param max_lon Maximum longitude value
 * @param min_lat Minimum latitude value
 * @param max_lat Maximum latitude value
 * @param num_lon Number of grid points in longitude direction
 * @param num_lat Number of grid points in latitude direction
 * @param lon_step Grid cell size in longitude direction
 * @param lat_step Grid cell size in latitude direction
 */
__global__ void cubicInterpolationKernel(
    const double* __restrict__ grid,
    const Point* __restrict__ points,
    double* __restrict__ results,
    int num_points,
    double min_lon, double max_lon,
    double min_lat, double max_lat,
    int num_lon, int num_lat,
    double lon_step, double lat_step
) {
    // Calculate the global thread ID
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (tid < num_points) {
        double lon = points[tid].lon;
        double lat = points[tid].lat;
        
        // Return NaN for out-of-bounds query points
        if (lon < min_lon || lon > max_lon || lat < min_lat || lat > max_lat) {
            results[tid] = NAN;
            return;
        }
        
        // Map (lon, lat) to grid (x, y) coordinates.
        double x = (lon - min_lon) / lon_step;
        double y = (lat - min_lat) / lat_step;
        int x_int = floor(x);
        int y_int = floor(y);
        double tx = x - x_int;
        double ty = y - y_int;

        // For cubic interpolation we need a 4x4 block of neighbors.
        double interpRows[4];  // Stores interpolation result for each of 4 rows.
        
        // Loop over four rows, offset by -1, 0, +1, +2 relative to y_int.
        for (int m = -1; m <= 2; m++) {
            int j = y_int + m;
            // Clamp to the valid row range.
            j = (j < 0) ? 0 : (j >= num_lat ? num_lat - 1 : j);
            
            double p[4];  // This holds four consecutive grid values in the row.
            // Loop over four columns, offset by -1, 0, +1, +2 relative to x_int.
            for (int n = -1; n <= 2; n++) {
                int i = x_int + n;
                // Clamp to the valid column range.
                i = (i < 0) ? 0 : (i >= num_lon ? num_lon - 1 : i);
                p[n + 1] = grid[j * num_lon + i];
            }
            // Interpolate along the x direction for this row.
            interpRows[m + 1] = cubicInterpolate(p[0], p[1], p[2], p[3], tx);
        }
        
        // Now perform the cubic interpolation along the y direction.
        results[tid] = cubicInterpolate(interpRows[0], interpRows[1], interpRows[2], interpRows[3], ty);
    }
}

// Define an inline device function for the variogram.
__device__ inline double variogram(double h, double sill, double range) {
    return sill * (1.0 - exp(-h / range));
}

/**
 * @brief CUDA kernel for ordinary kriging interpolation.
 * Uses 4 neighboring points and an exponential variogram model.
 */
__global__ void krigingInterpolationKernel(
    const double* __restrict__ grid,
    const Point* __restrict__ points,
    double* __restrict__ results,
    int num_points,
    double min_lon, double max_lon,
    double min_lat, double max_lat,
    int num_lon, int num_lat,
    double lon_step, double lat_step
) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < num_points) {
        double lon = points[tid].lon;
        double lat = points[tid].lat;
        if(lon < min_lon || lon > max_lon || lat < min_lat || lat > max_lat) {
            results[tid] = NAN;
            return;
        }
        
        double x = (lon - min_lon) / lon_step;
        double y = (lat - min_lat) / lat_step;
        int x0 = floor(x);
        int y0 = floor(y);
        int x1 = min(x0 + 1, num_lon - 1);
        int y1 = min(y0 + 1, num_lat - 1);
        
        // Get 4 neighbor elevations.
        double neighbors[4];
        neighbors[0] = grid[y0 * num_lon + x0];
        neighbors[1] = grid[y0 * num_lon + x1];
        neighbors[2] = grid[y1 * num_lon + x0];
        neighbors[3] = grid[y1 * num_lon + x1];
        
        // Compute neighbor coordinates.
        double coords[4][2];
        coords[0][0] = min_lon + x0 * lon_step;
        coords[0][1] = min_lat + y0 * lat_step;
        coords[1][0] = min_lon + x1 * lon_step;
        coords[1][1] = min_lat + y0 * lat_step;
        coords[2][0] = min_lon + x0 * lon_step;
        coords[2][1] = min_lat + y1 * lat_step;
        coords[3][0] = min_lon + x1 * lon_step;
        coords[3][1] = min_lat + y1 * lat_step;
        
        double q[2] = {lon, lat};
        
        // Variogram parameters.
        double sill = 100.0;
        double range = 10.0;
        
        // Build augmented 5x6 matrix (5 rows, 5 system coefficients, 6th column is right-hand side).
        double M[5][6] = {0};
        for (int i = 0; i < 5; i++) {
            for (int j = 0; j < 6; j++) {
                M[i][j] = 0.0;
            }
        }
        
        for (int i = 0; i < 4; i++) {
            for (int j = 0; j < 4; j++) {
                double dx = coords[i][0] - coords[j][0];
                double dy = coords[i][1] - coords[j][1];
                double d = sqrt(dx*dx + dy*dy);
                M[i][j] = variogram(d, sill, range);
            }
            M[i][4] = 1.0;
        }
        for (int j = 0; j < 4; j++) {
            M[4][j] = 1.0;
        }
        M[4][4] = 0.0;
        
        for (int i = 0; i < 4; i++) {
            double dx = coords[i][0] - q[0];
            double dy = coords[i][1] - q[1];
            double d = sqrt(dx*dx + dy*dy);
            M[i][5] = variogram(d, sill, range);
        }
        M[4][5] = 1.0;
        
        // Solve the 5x5 system via Gaussian elimination.
        const int N = 5;
        for (int i = 0; i < N; i++) {
            double pivot = M[i][i];
            if (fabs(pivot) < 1e-12) {
                results[tid] = NAN;
                return;
            }
            for (int j = i; j < N+1; j++) {
                M[i][j] /= pivot;
            }
            for (int k = 0; k < N; k++) {
                if (k == i) continue;
                double factor = M[k][i];
                for (int j = i; j < N+1; j++) {
                    M[k][j] -= factor * M[i][j];
                }
            }
        }
        double prediction = 0.0;
        for (int i = 0; i < 4; i++) {
            prediction += M[i][5] * neighbors[i];
        }
        results[tid] = prediction;
    }
}